#include "hip/hip_runtime.h"
#include "optimization.cuh"

#define BIAS 1.0f

using namespace CUDA;

// l = layer
// n = neuron number in layers
// w = weight number and result in neurons

extern "C"
{

//=====================================================================
//  /!\ WARINING /!\
//=====================================================================

// can still be optimized by decomposing the sum
// can still be optimized by using cuDNN for compute tanh(x)
// can still be optimized by suppressig all for loops
// Can we remove "int n = threadIdx.x;"?
// The best is no loop and no new variable
// Know if const take time or not
// use reference to primitive
// check that debugError produce 0 code in release mode
// do not reset error of output
// option compiler for MSVC
// compute brackpropagation in same time of calcul error ????
// view if use shared memory is more quick that pass pointer or value.
// see pragma unroll

//=====================================================================
//  The compute of output
//=====================================================================

__global__ void calculOutputPerceptronForInputLayer(float* weightsInInputLayer,
                                                    float* outputsInInputLayer,
                                                    const int numberOfInput,
                                                    float* inputs)
{
    int n = threadIdx.x;
    setOutputInInputLayer(outputsInInputLayer, n, 0.0f);
    #pragma unroll
    for(int w = 0; w < numberOfInput; w++)
    {
        addOutputInInputLayer(outputsInInputLayer, n, getWeightInInputLayer(weightsInInputLayer, numberOfInput, n, w) * inputs[w]);
    }
    setOutputInInputLayer(outputsInInputLayer, n, tanh(getOutputInInputLayer(outputsInInputLayer, n) + BIAS));
}

__global__ void calculOutputPerceptronForFistHiddenLayer(float* weightsInHiddenLayers,
                                                         float* outputsInInputLayer,
                                                         float* outputsInHiddenLayers,
                                                         const int numberOfNeuronInHiddenLayers)
{
    int n = threadIdx.x;
    setOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, n, 0.0f);
    #pragma unroll
    for(int w = 0; w < numberOfNeuronInHiddenLayers; w++)
    {
        addOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, n, getWeightInHiddenLayers(weightsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, n, w) *
                                getOutputInInputLayer(outputsInInputLayer, w));
    }
    setOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, n, tanh(getOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, n) + BIAS));
}

__global__ void calculOutputPerceptronForOtherHiddenLayers(float* weightsInHiddenLayers,
                                                           float* outputsInHiddenLayers,
                                                           const int numberOfNeuronInHiddenLayers,
                                                           const int l)
{
    int n = threadIdx.x;
    setOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, l, n, 0.0f);
    #pragma unroll
    for(int w = 0; w < numberOfNeuronInHiddenLayers; w++)
    {
        addOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, l, n, getWeightInHiddenLayers(weightsInHiddenLayers, numberOfNeuronInHiddenLayers, l, n, w)
                                * getOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, l-1, w));
    }
    setOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, l, n, tanh(getOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, l, n) + BIAS));
}

__global__ void calculOutputPerceptronForOutputLayer(float* weightsInOutputLayer,
                                                     float* outputsInHiddenLayers,
                                                     float* outputsInOutputLayer,
                                                     const int numberOfHiddenLayers,
                                                     const int numberOfNeuronInHiddenLayers)
{
    int n = threadIdx.x;
    setOutputInOutputLayer(outputsInOutputLayer, n, 0.0f);
    #pragma unroll
    for(int w = 0; w < numberOfNeuronInHiddenLayers; w++)
    {
        addOutputInOutputLayer(outputsInOutputLayer, n, getWeightInOutputLayer(weightsInOutputLayer, numberOfNeuronInHiddenLayers, n, w) *
                               getOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, numberOfHiddenLayers-1, w));
         //printf("w : %f\n", getOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, numberOfHiddenLayers-1, w));
    }
    //printf("output %i : %f\n", n, getOutputInOutputLayer(outputsInOutputLayer, n));
    setOutputInOutputLayer(outputsInOutputLayer, n, (tanh(getOutputInOutputLayer(outputsInOutputLayer, n) + BIAS))/2.0f + 0.5f);
}

__forceinline__ __host__ void calculOutputWithoutConvertInputs(float* weightsInInputLayer,
                                                               float* weightsInHiddenLayers,
                                                               float* weightsInOutputLayer,
                                                               float* outputsInInputLayer,
                                                               float* outputsInHiddenLayers,
                                                               float* outputsInOutputLayer,
                                                               const int numberOfHiddenLayers,
                                                               const int numberOfNeuronInHiddenLayers,
                                                               const int numberOfInput,
                                                               const int numberOfOutput,
                                                               float* inputs)
{
    calculOutputPerceptronForInputLayer<<<1, numberOfNeuronInHiddenLayers>>>(weightsInInputLayer,
                                                                             outputsInInputLayer,
                                                                             numberOfInput,
                                                                             inputs);
    debugError();

    calculOutputPerceptronForFistHiddenLayer<<<1, numberOfNeuronInHiddenLayers>>>(weightsInHiddenLayers,
                                                                                  outputsInInputLayer,
                                                                                  outputsInHiddenLayers,
                                                                                  numberOfNeuronInHiddenLayers);
    debugError();

    #pragma unroll
    for(int l = 2; l < numberOfHiddenLayers; l++)
    {
        calculOutputPerceptronForOtherHiddenLayers<<<1, numberOfNeuronInHiddenLayers>>>(weightsInHiddenLayers,
                                                                                        outputsInHiddenLayers,
                                                                                        numberOfNeuronInHiddenLayers,
                                                                                        l);
        debugError();
    }

    calculOutputPerceptronForOutputLayer<<<1, numberOfOutput>>>(weightsInOutputLayer,
                                                                outputsInHiddenLayers,
                                                                outputsInOutputLayer,
                                                                numberOfHiddenLayers,
                                                                numberOfNeuronInHiddenLayers);
    debugError();
}

__host__ void calculOutput(float* weightsInInputLayer,
                           float* weightsInHiddenLayers,
                           float* weightsInOutputLayer,
                           float* outputsInInputLayer,
                           float* outputsInHiddenLayers,
                           float* outputsInOutputLayer,
                           const int numberOfHiddenLayers,
                           const int numberOfNeuronInHiddenLayers,
                           const int numberOfInput,
                           const int numberOfOutput,
                           float* inputsCPU)
{
    float* inputs_device;
    hipMalloc((void**)&inputs_device, sizeof(float) * numberOfInput);
    hipMemcpy(inputs_device, inputsCPU, sizeof(float) * numberOfInput, hipMemcpyHostToDevice);

    calculOutputWithoutConvertInputs(weightsInInputLayer,
                                     weightsInHiddenLayers,
                                     weightsInOutputLayer,
                                     outputsInInputLayer,
                                     outputsInHiddenLayers,
                                     outputsInOutputLayer,
                                     numberOfHiddenLayers,
                                     numberOfNeuronInHiddenLayers,
                                     numberOfInput,
                                     numberOfOutput,
                                     inputs_device);


}



//=====================================================================
//  The compute of backpropagation
//=====================================================================

__global__ void resetErrorPerceptronForInputLayer(float* errorsInInputLayer)
{
    setErrorInInputLayer(errorsInInputLayer, threadIdx.x, 0.0f);
}

__global__ void resetErrorPerceptronForHiddenLayers(float* errorsInHiddenLayers,
                                                    const int numberOfNeuronInHiddenLayers,
                                                    const int l)
{
    setErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, l, threadIdx.x, 0.0f);
}

__global__ void resetErrorPerceptronForOutputLayer(float* errorsInOutputLayer)
{
    setErrorInOutputLayer(errorsInOutputLayer, threadIdx.x, 0.0f);
}

__forceinline__ __host__ void resetError(float* errorsInInputLayer,
                                         float* errorsInHiddenLayers,
                                         float* errorsInOutputLayer,
                                         const int numberOfHiddenLayers,
                                         const int numberOfNeuronInHiddenLayers,
                                         const int numberOfOutput)
{
    resetErrorPerceptronForOutputLayer<<<1, numberOfOutput>>>(errorsInOutputLayer);
    debugError();

    #pragma unroll
    for(int l = 0; l < numberOfHiddenLayers; l++)
    {
        resetErrorPerceptronForHiddenLayers<<<1, numberOfNeuronInHiddenLayers>>>(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, l);
        debugError();
    }

    resetErrorPerceptronForInputLayer<<<1, numberOfNeuronInHiddenLayers>>>(errorsInInputLayer);
    debugError();
}


__global__ void calculErrorPerceptronForOutputLayer(float* outputsInOutputLayer,
                                                    float* errorsInOutputLayer,
                                                    float* desires)
{
    int n = threadIdx.x;
    setErrorInOutputLayer(errorsInOutputLayer, n, (desires[n] - getOutputInOutputLayer(outputsInOutputLayer, n)) * abs(desires[n] - getOutputInOutputLayer(outputsInOutputLayer, n)));
    //printf("error GPU (%i,%i) : %f\n", 10, n, getErrorInOutputLayer(errorsInOutputLayer, n));
}

__global__ void calculErrorPerceptronForLastHiddenLayers(float* weightsInOutputLayer,
                                                         float* errorsInHiddenLayers,
                                                         float* errorsInOutputLayer,
                                                         const int numberOfHiddenLayers,
                                                         const int numberOfOutput,
                                                         const int numberOfNeuronInHiddenLayers)
{
    int n = threadIdx.x;
    #pragma unroll
    for(int w = 0; w < numberOfOutput; w++)
    {
        addErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, numberOfHiddenLayers-1, n,
                               getErrorInOutputLayer(errorsInOutputLayer, w) *
                               getWeightInOutputLayer(weightsInOutputLayer, numberOfNeuronInHiddenLayers, w, n));
    }
    divideErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, numberOfHiddenLayers-1, n, numberOfOutput);
    //printf("error GPU (%i,%i) : %f\n", numberOfHiddenLayers-1, n, getErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, numberOfHiddenLayers-1, n));
}

__global__ void calculErrorPerceptronForOtherHiddenLayers(float* weightsInHiddenLayers,
                                                          float* errorsInHiddenLayers,
                                                          const int numberOfNeuronInHiddenLayers,
                                                          const int l)
{
    int n = threadIdx.x;
    #pragma unroll
    for(int w = 0; w < numberOfNeuronInHiddenLayers; w++)
    {
        addErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, l, n,
                               getErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, l+1, w) *
                               getWeightInHiddenLayers(weightsInHiddenLayers, numberOfNeuronInHiddenLayers, l+1, w, n));

        //if(n == 0 && w == 3)
        //    printf("error GPU (%i,%i) : %f\n", l+1, w, getErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, l+1, w));
    }
    divideErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, l, n, numberOfNeuronInHiddenLayers);
    //printf("error GPU : %f\n", getErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, n));
}

__global__ void calculErrorPerceptronForInputLayer(float* weightsInHiddenLayers,
                                                   float* errorsInInputLayer,
                                                   float* errorsInHiddenLayers,
                                                   const int numberOfNeuronInHiddenLayers)
{
    int n = threadIdx.x;
    #pragma unroll
    for(int w = 0; w < numberOfNeuronInHiddenLayers; w++)
    {
         addErrorInInputLayer(errorsInInputLayer, n,
                              getErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, w) *
                              getWeightInHiddenLayers(weightsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, w, n));
    }
    divideErrorInInputLayer(errorsInInputLayer, n, numberOfNeuronInHiddenLayers);
    //printf("error GPU il : %f\n", getErrorInInputLayer(errorsInInputLayer, n));
}

__global__ void trainPerceptronForOutputLayer(float* weightsInOutputLayer,
                                              float* outputsInHiddenLayers,
                                              float* errorsInOutputLayer,
                                              const int numberOfHiddenLayers,
                                              const int numberOfNeuronInHiddenLayers,
                                              float learningRate)
{
    int n = threadIdx.x;
    #pragma unroll
    for(int w = 0;w < numberOfNeuronInHiddenLayers; w++) // weights[i] += learningRate * error * inputs[i];
    {
        addWeightInOutputLayer(weightsInOutputLayer, numberOfNeuronInHiddenLayers, n, w,
                               learningRate *
                               getErrorInOutputLayer(errorsInOutputLayer, n) *
                               getOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, numberOfHiddenLayers-1, w));
    }
}

__global__ void trainPerceptronForOtherHiddenLayer(float* weightsInHiddenLayers,
                                                   float* outputsInHiddenLayers,
                                                   float* errorsInHiddenLayers,
                                                   const int numberOfNeuronInHiddenLayers,
                                                   const int l,
                                                   float learningRate)
{
    int n = threadIdx.x;
    #pragma unroll
    for(int w = 0; w < numberOfNeuronInHiddenLayers; w++) // weights[i] += learningRate * error * inputs[i];
    {
        addWeightInHiddenLayers(weightsInHiddenLayers, numberOfNeuronInHiddenLayers, l, n, w,
                                learningRate *
                                getErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, l, n) *
                                getOutputInHiddenLayers(outputsInHiddenLayers, numberOfNeuronInHiddenLayers, l-1, w));
    }
}

__global__ void trainPerceptronForFirtHiddenLayer(float* weightsInHiddenLayers,
                                                  float* outputsInInputLayer,
                                                  float* errorsInHiddenLayers,
                                                  const int numberOfNeuronInHiddenLayers,
                                                  float learningRate)
{
    int n = threadIdx.x;
    #pragma unroll
    for(int w = 0; w < numberOfNeuronInHiddenLayers; w++) // weights[i] += learningRate * error * inputs[i];
    {
        addWeightInHiddenLayers(weightsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, n, w,
                                learningRate *
                                getErrorInHiddenLayers(errorsInHiddenLayers, numberOfNeuronInHiddenLayers, 1, n) *
                                getOutputInInputLayer(outputsInInputLayer, w));
    }
}

__global__ void trainPerceptronForInputLayer(float* weightsInInputLayer,
                                             float* errorsInInputLayer,
                                             const int numberOfInput,
                                             float learningRate,
                                             float* inputs)
{
    int n = threadIdx.x;
    #pragma unroll
    for(int w = 0; w < numberOfInput; w++) // weights[i] += learningRate * error * inputs[i];
    {
        addWeightInInputLayer(weightsInInputLayer, numberOfInput, n, w,
                              learningRate *
                              getErrorInInputLayer(errorsInInputLayer, n) *
                              inputs[w]);
    }
}

void backpropagation(float* weightsInInputLayer,
                     float* weightsInHiddenLayers,
                     float* weightsInOutputLayer,
                     float* outputsInInputLayer,
                     float* outputsInHiddenLayers,
                     float* outputsInOutputLayer,
                     float* errorsInInputLayer,
                     float* errorsInHiddenLayers,
                     float* errorsInOutputLayer,
                     const int numberOfHiddenLayers,
                     const int numberOfNeuronInHiddenLayers,
                     const int numberOfInput,
                     const int numberOfOutput,
                     float learningRate,
                     float* inputsCPU,
                     float* desiresCPU)
{
    float* inputsDevice;
    hipMalloc((void**)&inputsDevice, sizeof(float) * numberOfInput);
    hipMemcpy(inputsDevice, inputsCPU, sizeof(float) * numberOfInput, hipMemcpyHostToDevice);

    float* desiresDevice;
    hipMalloc((void**)&desiresDevice, sizeof(float) * numberOfInput);
    hipMemcpy(desiresDevice, desiresCPU, sizeof(float) * numberOfInput, hipMemcpyHostToDevice);

    calculOutputWithoutConvertInputs(weightsInInputLayer,
                                     weightsInHiddenLayers,
                                     weightsInOutputLayer,
                                     outputsInInputLayer,
                                     outputsInHiddenLayers,
                                     outputsInOutputLayer,
                                     numberOfHiddenLayers,
                                     numberOfNeuronInHiddenLayers,
                                     numberOfInput,
                                     numberOfOutput,
                                     inputsDevice);

    resetError(errorsInInputLayer,
               errorsInHiddenLayers,
               errorsInOutputLayer,
               numberOfHiddenLayers,
               numberOfNeuronInHiddenLayers,
               numberOfOutput);

    calculErrorPerceptronForOutputLayer<<<1, numberOfOutput>>>(outputsInOutputLayer,
                                                               errorsInOutputLayer,
                                                               desiresDevice);
    debugError();

    calculErrorPerceptronForLastHiddenLayers<<<1, numberOfNeuronInHiddenLayers>>>(weightsInOutputLayer,
                                                                                  errorsInHiddenLayers,
                                                                                  errorsInOutputLayer,
                                                                                  numberOfHiddenLayers,
                                                                                  numberOfOutput,
                                                                                  numberOfNeuronInHiddenLayers);
    debugError();

    #pragma unroll
    for(int l = numberOfHiddenLayers-2; l >= 1; l--) // show - 1
    {
        calculErrorPerceptronForOtherHiddenLayers<<<1, numberOfNeuronInHiddenLayers>>>(weightsInHiddenLayers,
                                                                                       errorsInHiddenLayers,
                                                                                       numberOfNeuronInHiddenLayers,
                                                                                       l);
        debugError();
    }

    calculErrorPerceptronForInputLayer<<<1, numberOfNeuronInHiddenLayers>>>(weightsInHiddenLayers,
                                                                            errorsInInputLayer,
                                                                            errorsInHiddenLayers,
                                                                            numberOfNeuronInHiddenLayers);
    debugError();

    trainPerceptronForOutputLayer<<<1, numberOfOutput>>>(weightsInOutputLayer,
                                                         outputsInHiddenLayers,
                                                         errorsInOutputLayer,
                                                         numberOfHiddenLayers,
                                                         numberOfNeuronInHiddenLayers,
                                                         learningRate);
    debugError();

    #pragma unroll
    for(int l = numberOfHiddenLayers-1; l >= 2; l--)
    {
        trainPerceptronForOtherHiddenLayer<<<1, numberOfNeuronInHiddenLayers>>>(weightsInHiddenLayers,
                                                                                outputsInHiddenLayers,
                                                                                errorsInHiddenLayers,
                                                                                numberOfNeuronInHiddenLayers,
                                                                                l,
                                                                                learningRate);
        debugError();
    }

    trainPerceptronForFirtHiddenLayer<<<1, numberOfNeuronInHiddenLayers>>>(weightsInHiddenLayers,
                                                                           outputsInInputLayer,
                                                                           errorsInHiddenLayers,
                                                                           numberOfNeuronInHiddenLayers,
                                                                           learningRate);
    debugError();

    trainPerceptronForInputLayer<<<1, numberOfNeuronInHiddenLayers>>>(weightsInInputLayer,
                                                                      errorsInInputLayer,
                                                                      numberOfInput,
                                                                      learningRate,
                                                                      inputsDevice);
    debugError();
}

//=====================================================================
//  The compute for all data
//=====================================================================


__forceinline__ float calculateClusteringRate(float* weightsInInputLayer,
                                              float* weightsInHiddenLayers,
                                              float* weightsInOutputLayer,
                                              float* outputsInInputLayer,
                                              float* outputsInHiddenLayers,
                                              float* outputsInOutputLayer,
                                              const int numberOfHiddenLayers,
                                              const int numberOfNeuronInHiddenLayers,
                                              const int numberOfInput,
                                              const int numberOfOutput,
                                              float* inputs,
                                              float* desires,
                                              const int sizeOfTestingSet)
{
    int numberOfResultsClassifiedWell;
    int numberOfResultsMisclassefied;

    for(int i = 0; i < sizeOfTestingSet; i++)
    {
        calculOutputWithoutConvertInputs(weightsInInputLayer,
                                         weightsInHiddenLayers,
                                         weightsInOutputLayer,
                                         outputsInInputLayer,
                                         outputsInHiddenLayers,
                                         outputsInOutputLayer,
                                         numberOfHiddenLayers,
                                         numberOfNeuronInHiddenLayers,
                                         numberOfInput,
                                         numberOfOutput,
                                         &inputs[numberOfInput*i]);

        for(int j = 0; j < numberOfOutput; j++)// EXECUTE INSIDE GLOBAL DESIRES IN IN GPU
        {
            if(desires[j] == 0 && outputsInOutputLayer[j] >= 0.5f
            || desires[j] == 1 && outputsInOutputLayer[j] < 0.5f)
            {
                numberOfResultsMisclassefied ++;
                break;
            }
            else if(j == numberOfOutput-1)
            {
               numberOfResultsClassifiedWell ++;
               break;
            }
        }
    }
    return -2.0;//(float)numberOfResultsClassifiedWell/(numberOfResultsClassifiedWell+numberOfResultsMisclassefied);
}

__forceinline__ void shuffleLearningSet(float* array,
                                        const int sizeOfLearningSet,
                                        int random,
                                        float temp)
{
    for(int i = 0; i < sizeOfLearningSet; i++)
    {
        random = rand()%sizeOfLearningSet;
        temp = array[random];
        array[random] = array[i];
        array[i] = temp;
    }
}

__forceinline__ void saveOnCPU()
{

}

void TrainWithAllDatabase(float* weightsInInputLayer,
                          float* weightsInHiddenLayers,
                          float* weightsInOutputLayer,
                          float* outputsInInputLayer,
                          float* outputsInHiddenLayers,
                          float* outputsInOutputLayer,
                          float* errorsInInputLayer,
                          float* errorsInHiddenLayers,
                          float* errorsInOutputLayer,
                          const int numberOfHiddenLayers,
                          const int numberOfNeuronInHiddenLayers,
                          const int numberOfInput,
                          const int numberOfOutput,
                          float learningRate,
                          float* inputsLearningCPU,
                          float* inputsTestingCPU,
                          float* desiresLearningCPU,
                          float* desiresTestingCPU,
                          const int sizeOfLearningSet,
                          const int sizeOfTestingSet,
                          float clusteringRate,
                          const int numberOfTrain,
                          const int frequencyOfClusteringRateVerification)
{
    //float newClusteringRate = 0;
    float* inputsLearningDevice;
    float* desiresLearningDevice;
    float* inputsTestingDevice;
    float* desiresTestingDevice;

    srand(time(NULL));

    hipMalloc((void**)&inputsLearningDevice, sizeof(float) * numberOfInput * sizeOfLearningSet);
    hipMemcpy(inputsLearningDevice, inputsLearningCPU, sizeof(float) * numberOfInput * sizeOfLearningSet, hipMemcpyHostToDevice);

    hipMalloc((void**)&desiresLearningDevice, sizeof(float) * numberOfInput* sizeOfLearningSet);
    hipMemcpy(desiresLearningDevice, desiresLearningCPU, sizeof(float) * numberOfInput * sizeOfLearningSet, hipMemcpyHostToDevice);

    hipMalloc((void**)&inputsTestingDevice, sizeof(float) * numberOfInput * sizeOfTestingSet);
    hipMemcpy(inputsTestingDevice, inputsTestingCPU, sizeof(float) * numberOfInput * sizeOfTestingSet, hipMemcpyHostToDevice);

    hipMalloc((void**)&desiresTestingDevice, sizeof(float) * numberOfInput * sizeOfTestingSet);
    hipMemcpy(desiresTestingDevice, desiresTestingCPU, sizeof(float) * numberOfInput * sizeOfTestingSet, hipMemcpyHostToDevice);

    int i, j, index;

    int randomForShuffle;
    float tempForShuffle;
    float* arrayForShuffle = (float*)malloc(sizeof(float) * sizeOfLearningSet);
    for(int i = 0; i < sizeOfLearningSet; i++)
    {
        arrayForShuffle[i] = i;
    }

    for(i = 0; i < numberOfTrain;)
    {
        for(j = 0; j < frequencyOfClusteringRateVerification; i++, j++)
        {
            if(i%sizeOfLearningSet == 0)
            {
                shuffleLearningSet(arrayForShuffle,
                                   sizeOfLearningSet,
                                   randomForShuffle,
                                   tempForShuffle);
            }
            index = arrayForShuffle[i%sizeOfLearningSet];

            backpropagation(weightsInInputLayer,
                            weightsInHiddenLayers,
                            weightsInOutputLayer,
                            outputsInInputLayer,
                            outputsInHiddenLayers,
                            outputsInOutputLayer,
                            errorsInInputLayer,
                            errorsInHiddenLayers,
                            errorsInOutputLayer,
                            numberOfHiddenLayers,
                            numberOfNeuronInHiddenLayers,
                            numberOfInput,
                            numberOfOutput,
                            learningRate,
                            &inputsLearningDevice[index*numberOfInput],
                            &desiresLearningDevice[index*numberOfOutput]);
        }

        calculateClusteringRate(weightsInInputLayer,
                                weightsInHiddenLayers,
                                weightsInOutputLayer,
                                outputsInInputLayer,
                                outputsInHiddenLayers,
                                outputsInOutputLayer,
                                numberOfHiddenLayers,
                                numberOfNeuronInHiddenLayers,
                                numberOfInput,
                                numberOfOutput,
                                &inputsTestingDevice[0],
                                &desiresTestingDevice[0],
                                sizeOfTestingSet);

        /*if(newClusteringRate > clusteringRate)
        {
            clusteringRate = newClusteringRate;
            saveOnCPU();
            printf("Clustering rate = %f", clusteringRate);
        }*/
    }
}


//=====================================================================
//  Tool functions
//=====================================================================

__host__ void returnNetworkOnCPU(float* weightsInInputLayerCPU,
                                 float* weightsInHiddenLayersCPU,
                                 float* weightsInOutputLayerCPU,
                                 float* weightsInInputLayerGPU,
                                 float* weightsInHiddenLayersGPU,
                                 float* weightsInOutputLayerGPU,
                                 int numberOfHiddenLayers,
                                 int numberOfNeuronsInHiddenLayers,
                                 int numberOfInput,
                                 int numberOfOutput)
{
    hipMemcpy(weightsInInputLayerCPU, weightsInInputLayerGPU, sizeof(float) * numberOfHiddenLayers * numberOfInput ,hipMemcpyDeviceToHost);
    hipMemcpy(weightsInHiddenLayersCPU, weightsInHiddenLayersGPU, sizeof(float) * numberOfNeuronsInHiddenLayers * numberOfNeuronsInHiddenLayers * (numberOfHiddenLayers-1), hipMemcpyDeviceToHost);
    hipMemcpy(weightsInOutputLayerCPU, weightsInOutputLayerGPU, sizeof(float) * numberOfOutput * numberOfNeuronsInHiddenLayers, hipMemcpyDeviceToHost);
}

__host__ void returnOutputOnCPU(float* outputsInOutputLayerCPU,
                                float* outputsInOutputLayerGPU,
                                int numberOfOutput)
{
    hipMemcpy(outputsInOutputLayerCPU, outputsInOutputLayerGPU, sizeof(float) * numberOfOutput, hipMemcpyDeviceToHost);
    debugError();
}


}
